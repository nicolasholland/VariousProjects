
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <stdlib.h>

__global__
void kmeanspredict(int n, int nofc, float *x, float *y, float* model, float* label)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i >= n)
  	return;

  int iter;
  double dist = 9999;
  double tmp = 0;
  double res = 2;

  for (iter = 0; iter < nofc; iter++) {
  	tmp = sqrt((x[i] - model[iter]) * (x[i] - model[iter]) +
  			   (y[i] - model[iter + nofc]) * (y[i] - model[iter + nofc]));
  	res = tmp < dist ? iter : res;
  	dist = tmp < dist ? tmp : dist;
  }
  label[i] = res;
}

int main(void)
{
  int N = 100000000;
  float *feature1, *feature2, *d_feature1, *d_feature2, *d_model;
  float *label, *d_label;
  feature1 = (float*)malloc(N*sizeof(float));
  feature2 = (float*)malloc(N*sizeof(float));
  label = (float*)malloc(N*sizeof(int));

  hipMalloc(&d_feature1, N*sizeof(float));
  hipMalloc(&d_feature2, N*sizeof(float));
  hipMalloc(&d_label, N*sizeof(float));

  float model[] = {0.79314066 , 0.40563098, 0.27847279,
                   0.27847279, 0.8073302, 0.28528738};
  hipMalloc(&d_model, 6*sizeof(float));

  std::srand(time(NULL));

  for (int i = 0; i < N; i++) {
	feature1[i] = rand();
	feature2[i] = rand();
  }

  clock_t begin = std::clock();

  hipMemcpy(d_feature1, feature1, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_feature2, feature2, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_model, model, 6*sizeof(float), hipMemcpyHostToDevice);


  kmeanspredict<<<(N+255)/256, 256>>>(N, 3, d_feature1, d_feature2, d_model, d_label);


  hipMemcpy(feature2, d_feature2, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(label, d_label, N*sizeof(float), hipMemcpyDeviceToHost);

  clock_t end = std::clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("measured time: %lf\n", elapsed_secs);

  hipFree(d_feature1);
  hipFree(d_feature2);
  hipFree(d_label);
  free(feature1);
  free(feature2);
  free(label);
}
